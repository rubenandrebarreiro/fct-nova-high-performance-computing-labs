#include "hip/hip_runtime.h"
/**
 *
 * High Performance Computing
 * Practical Lab #1
 * Exercise 1)
 *
 * Description/Steps of operations performed by the GPU CUDA's Kernels:
 * - 1st) The sum of the content of two Arrays of Integers,
 *        resulting on a third Array of Integers,
 *        using CUDA-based parallel operations in the GPU.
 *
 * - 2nd) The decrement of the content of the third Array of Integers,
 *        using CUDA-based parallel operations in the GPU.
 *
 * - 3rd) The reverse of the content of the third Array of Integers,
 *        resulting on a fourth Array of Integers,
 *        using CUDA-based parallel operations in the GPU.
 *
 *   Implementation in C.
 *
 * Authors:
 * - Herve Miguel Paulino (Professor)
 * - Ruben Andre Barreiro
 *
 */

#include "../../../../include/cadlabs.hpp"
#include "../../../../include/timer.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <stdlib.h>
#include <stdio.h>

#define THREADS_PER_BLOCK (512)
#define BLOCK_DIMENSIONS (1024 * 1024)


using namespace std;

/**
 *
 * Simple utility function to check for runtime errors in GPU CUDA's Kernels.
 *
 * @param msg the message related to the runtime errors in GPU CUDA's Kernels
 *
 */
void checkCUDAError(const char* msg);

/**
 *
 * 1st step of operations performed by the GPU CUDA's Kernels:
 *
 * The GPU CUDA's Kernel to perform a sum of all the two Integers contained in two Arrays,
 * performing the computing in parallel.
 *
 * @param a a pointer to the 1st Array of Integers
 * @param b a pointer to the 2nd Array of Integers
 * @param c a pointer to the 3rd Array of Integers
 *          for the final result of the operation
 * @param size_array the total size of the Arrays of Integers, previously allocated
 *
 */
__global__ void addKernel(const int* a, const int* b, int* c, const unsigned int size_array) {

    // Index of Kernel/Thread = (Number of Blocks x Size of Block) + Offset associated to the Thread,
    //                           that will perform the Action
    int indexKernelThread = (blockIdx.x * blockDim.x) + threadIdx.x;

    // To guarantee that not all the Threads will be perform the action, since that for sometimes,
    // it's necessary more Threads than filled positions in the Arrays of Integers
    if(indexKernelThread < size_array) {
        c[indexKernelThread] = a[indexKernelThread] + b[indexKernelThread];
    }

    // Block until all Threads in the block have written their data to shared memory
    // (a Threads' barrier - not necessary, in this case)
    //__syncthreads();
}

/**
 *
 * 2nd step of operations performed by the GPU CUDA's Kernels:
 *
 * The GPU CUDA's Kernel to perform a decrement of an Array of Integers, previously calculated by
 * a sum of all the two Integers contained in two Arrays, performing the computing in parallel.
 *
 * @param c a pointer to the 3rd Array of Integers,
 *          where will be decrement the Integer value contained in each position of the Array
 * @param size_array the total size of the Arrays of Integers, previously allocated
 *
 */
__global__ void decKernel(int* c, const unsigned int size_array) {

    // Index of Kernel/Thread = (Number of Blocks x Size of Block) + Offset associated to the Thread,
    //                           that will perform the Action
    int indexKernelThread = (blockIdx.x * blockDim.x) + threadIdx.x;

    // To guarantee that not all the Threads will be perform the action, since that for sometimes,
    // it's necessary more Threads than filled positions in the Arrays of Integers
    if(indexKernelThread < size_array) {
        c[indexKernelThread]--;
    }

    // Block until all Threads in the block have written their data to shared memory
    // (a Threads' barrier - not necessary, in this case)
    //__syncthreads();
}

/**
 *
 * 3rd step of operations performed by the GPU CUDA's Kernels:
 *
 * The GPU CUDA's Kernel to perform a reversal of an Array of Integers, previously calculated by
 * a decrement of all the Integers contained in an Array, performing the computing in parallel.
 *
 * @param c a pointer to the 3rd Array of Integers
 * @param d a pointer to the 4th Array of Integers,
 *          where will be reversed all the Integer values' positions contained in
 *          each position of the 3rd Array of Integers
 * @param size_array the total size/number of positions of the Arrays of Integers, previously allocated
 *
 */
__global__ void reverseKernel(const int* c, int* d, const unsigned int size_array) {

    // Index of Kernel/Thread = (Number of Blocks x Size of Block) + Offset associated to the Thread,
    //                                  that will perform the Action
    int indexKernelThread = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(indexKernelThread < (size_array / 2)) {

        // Perform the necessary exchanges between the positions of the Array of Integers,
        // in order to reverse all the content of the
        d[size_array - indexKernelThread - 1] = c[indexKernelThread];
        d[indexKernelThread] = c[size_array - indexKernelThread - 1];
    }
    else if (indexKernelThread == (size_array / 2)) {

        // In the case of the size of the Array of Integers is an odd number
        if((size_array % 2) > 0) {

            // It's not necessary to exchange the position of the middle of the Array of Integers,
            // the content of this position will be the same in both Array of Integers
            d[indexKernelThread] = c[indexKernelThread];
        }
    }

    // Block until all Threads in the block have written their data to shared memory
    // (a Threads' barrier - not necessary, in this case)
    //__syncthreads();
}

/**
 * The Main method to perform all the operations by the GPU CUDA's Kernels.
 *
 * @return 0
 */
int main() {

    // Size of the Arrays will be for 4 Integers
    constexpr auto size_array = 4;

    // Size in bytes for the Arrays of Integers
    constexpr auto size_in_bytes = size_array * sizeof(int);

    // Start to calculate all the computing time
    cadlabs::timer<> GPUComputingTimer;
    GPUComputingTimer.start();

    // Initialize random seed to generate random integers
    srand(time(NULL));

    // The CPU/RAM memory buffers in the Host for the 4 Arrays of Integers
    // (2 Arrays for the Inputs and 2 Arrays for the Outputs)
    int* host_a;
    int* host_b;
    int* host_c;
    int* host_d;

    // The GPU memory buffers in the Device for the 4 Arrays of Integers
    // (2 Arrays for the Inputs and 2 Arrays for the Outputs)
    int* dev_a;
    int* dev_b;
    int* dev_c;
    int* dev_d;

    // Allocate CPU/RAM memory buffers in the Host for the 4 Arrays of Integers
    // (2 Arrays for the Inputs and 2 Array for the Output)
    host_a = (int*) malloc(size_in_bytes); // The Array for the Input no. 1 (Array A)
    host_b = (int*) malloc(size_in_bytes); // The Array for the Input no. 2 (Array B)
    host_c = (int*) malloc(size_in_bytes); // The Array for the Output no. 1 (Array C)
    host_d = (int*) malloc(size_in_bytes); // The Array for the Output no. 2 (Array D)

    // Allocate GPU memory buffers in the Device for the 4 Arrays of Integers
    // (2 Arrays for the Inputs and 2 Array for the Output)
    hipMalloc((void **)&dev_a, size_in_bytes); // The Array for the Input no. 1 (Array A)
    hipMalloc((void **)&dev_b, size_in_bytes); // The Array for the Input no. 2 (Array B)
    hipMalloc((void **)&dev_c, size_in_bytes); // The Array for the Output no. 1 (Array C)
    hipMalloc((void **)&dev_d, size_in_bytes); // The Array for the Output no. 2 (Array D)

    // Check if allocation of GPU memory buffers in the device generated an error
    checkCUDAError("Error in allocation of Devices' memory");

    // Initialising the 1st Array of Integers for the Inputs with random values
    for(int i = 0; i < size_array; i++) {
        host_a[i] = (rand() % 5000);
    }

    // Initialising the 2nd Array of Integers for the Inputs with random values
    for(int i = 0; i < size_array; i++) {
        host_b[i] = (rand() % 5000);
    }

    // Just for debug of the Arrays of Integers for the Inputs
    printf("A = [");
    for(unsigned int i = 0; i < size_array - 1; i++) {
        printf("%d, ", host_a[i]);
    }
    printf("%d]\n", host_a[size_array - 1]);

    printf("B = [");
    for(unsigned int i = 0; i < size_array - 1; i++) {
        printf("%d, ", host_b[i]);
    }
    printf("%d]\n", host_b[size_array - 1]);

    // Copy the Arrays of Integers for the Inputs (Arrays A and B) from the
    // CPU/RAM memory buffers in the Host to the GPU memory buffers of the Device
    hipMemcpy(dev_a, host_a, size_in_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, size_in_bytes, hipMemcpyHostToDevice);

    // Check if the copy memory of the Host to the memory of the Device generated an error
    checkCUDAError("Error in copy memory of the Host to the memory of the Device");

    // Perform the Sum operation
    addKernel<<< (BLOCK_DIMENSIONS / THREADS_PER_BLOCK),
                  THREADS_PER_BLOCK >>>(dev_a, dev_b, dev_c, size_array);

    // Wait for the kernel to finish,
    // and return any errors encountered during the launch
    hipDeviceSynchronize();

    // Check if kernel execution generated an error
    checkCUDAError("Error in Kernel's invocation");

    // Copy the Array of Integers for the Output (Array C) from the
    // GPU memory buffers of the Device to the CPU/RAM memory buffers of the Host
    hipMemcpy(host_c, dev_c, size_in_bytes, hipMemcpyDeviceToHost);

    // Check if the copy memory of the Device to the memory of the Host generated an error
    checkCUDAError("Error in copy memory of the Device to the memory of the Host");

    // Just for debug of the 3rd Array of Integers
    printf("C = A + B = [");
    for(unsigned int i = 0; i < size_array - 1; i++) {
        printf("%d, ", host_c[i]);
    }
    printf("%d]\n", host_c[size_array - 1]);

    // Perform the Decrement operation
    decKernel<<< (BLOCK_DIMENSIONS / THREADS_PER_BLOCK),
                  THREADS_PER_BLOCK >>>(dev_c, size_array);

    // Wait for the kernel to finish,
    // and return any errors encountered during the launch
    hipDeviceSynchronize();

    // Copy the Array of Integers for the Output (Array C) from the
    // GPU memory buffers of the Device to the CPU/RAM memory buffers of the Host
    hipMemcpy(host_c, dev_c, size_in_bytes, hipMemcpyDeviceToHost);

    // Check if the copy memory of the Device to the memory of the Host generated an error
    checkCUDAError("Error in copy memory of the Device to the memory of the Host");

    // Just for debug of the 3rd Array of Integers
    printf("C - 1 = [");
    for(unsigned int i = 0; i < size_array - 1; i++) {
        printf("%d, ", host_c[i]);
    }
    printf("%d]\n", host_c[size_array - 1]);

    // Perform the Reverse operation
    reverseKernel<<< ((BLOCK_DIMENSIONS / THREADS_PER_BLOCK) / 2),
                       THREADS_PER_BLOCK >>>(dev_c, dev_d, size_array);

    // Wait for the kernel to finish,
    // and return any errors encountered during the launch
    hipDeviceSynchronize();

    // Copy the Array of Integers for the Output (Array D) from the
    // GPU memory buffers of the Device to the CPU/RAM memory buffers of the Host
    hipMemcpy(host_d, dev_d, size_in_bytes, hipMemcpyDeviceToHost);

    // Check if the copy memory of the Device to the memory of the Host generated an error
    checkCUDAError("Error in copy memory of the Device to the memory of the Host");

    // Just for debug of the 4th Array of Integers
    printf("D (Reversal of C) = [");
    for(unsigned int i = 0; i < size_array - 1; i++) {
        printf("%d, ", host_d[i]);
    }
    printf("%d]\n", host_d[size_array - 1]);

    // Free CPU/RAM memory buffers for the 4 Arrays of Integers,
    // previously allocated in Host memory
    // (2 Arrays for the Inputs and 2 Array for the Output)
    free(host_a);
    free(host_b);
    free(host_c);
    free(host_d);

    // Free GPU memory buffers for the 4 Arrays of Integers,
    // previously allocated in Device memory
    // (2 Arrays for the Inputs and 2 Arrays for the Output)
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_d);

    // Stop to calculate all the computing time
    GPUComputingTimer.stop();

    // Print the metrics of the all computing time
    GPUComputingTimer.print_stats(cout);
    cout << " milliseconds\n ";

    return 0;
}

/**
 *
 * Simple utility function to check for runtime errors in GPU CUDA's Kernels.
 *
 * @param msg the message related to the runtime errors in GPU CUDA's Kernel
 *
 */
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();

    if(hipSuccess != err) {
        fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err));

        exit(EXIT_FAILURE);
    }
}